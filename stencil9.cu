
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//#define abs(a) ((a)>0?(a):-(a))
//#define min(a,b) ((a>b)?(b):(a))
#ifndef SIZEX
#define SIZEX 10000
#endif
#ifndef SIZEY
#define SIZEY 10000
#endif
#ifndef TILEX
#define TILEX 50
#endif
#ifndef TILEY
#define TILEY 50
#endif
#include <omp.h>


void dump_stencil(float *a, FILE* f) {
	int i, j ;
	for (i=0; i < SIZEX; i++) {
	for (j=0; j < SIZEY;j++) {
		fprintf(f, "%lf ", a[i*SIZEY+j]) ;
	}
	fprintf(f, "\n");
	}
}

__global__ void stencil9(float *a, const float *b) {
  int i = threadIdx.x + blockIdx.x * blockDim.x ;
  int j = threadIdx.y + blockIdx.y * blockDim.y ;
  if (i >= 2 && j >= 2 && i < SIZEX-2 && j < SIZEY-2)
      a[i*SIZEY+j] = (8*b[i*SIZEY+j] + b[(i+1)*SIZEY+j] + b[(i+2)*SIZEY+j] 
			+ b[(i-1)*SIZEY+j] + b[(i-2)*SIZEY+j] + b[i*SIZEY+j+1]
			+ b[i*SIZEY+j-1] + b[i*SIZEY+j-2] + b[i*SIZEY+j+2])/9.;
}
float dot1D(float *a,float *b,int n)  {
  int i;
  float s=0;
  for (i=2; i<n-2; i++) 
    s+=a[i]*b[i];
  return s;
}
float dot2D(float *a,float *b)  {
  int i;
  float s=0;
  for (i=2; i<SIZEX-2; i++) 
    s+=dot1D(&a[i*SIZEY],&b[i*SIZEY],SIZEY);
  return s;
}
 int main() {
   int i,j,k;
  float *a,*b;
  float s=0;
  a=(float *)malloc(sizeof(float)*SIZEX*SIZEY);
  b=(float*)malloc(sizeof(float)*SIZEX*SIZEY);
  /* Initialization */
  for (i=0; i<SIZEX; i++)    
  for (j=0; j<SIZEY; j++)    
    a[i*SIZEY+j]=b[i*SIZEY+j]=0;
    for (j=SIZEY/4; j<SIZEY/2; j++) 
  for (i=SIZEX/4; i<SIZEX/2; i++) 
      b[i*SIZEY+j]=a[i*SIZEY+j]=1;
  /* Iterative computation. Iterate while error greater than ERROR */

  float *a_gpu, *b_gpu ;
  hipMalloc((void**) &a_gpu, SIZEX*SIZEY*sizeof(float)) ;
  hipMalloc((void**) &b_gpu, SIZEX*SIZEY*sizeof(float)) ;
  hipMemcpy(a_gpu, a, SIZEX*SIZEY*sizeof(float), hipMemcpyHostToDevice) ;
  hipMemcpy(b_gpu, b, SIZEX*SIZEY*sizeof(float), hipMemcpyHostToDevice) ;
  
  hipEvent_t start, stop ;
  hipEventCreate(&start) ;
  hipEventCreate(&stop) ;
  hipEventRecord(start);
  dim3 grid(SIZEX/TILEX, SIZEY/TILEY) ;
  dim3 block(TILEX, TILEY) ;
  for(k=0;k<100;k++) {
          
    stencil9<<<grid, block>>>(a_gpu,b_gpu);
    hipDeviceSynchronize() ;
    stencil9<<<grid, block>>>(b_gpu,a_gpu);
    hipDeviceSynchronize() ;
    fprintf(stderr,".");
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  fprintf(stderr,"\n%f\n",s);
  float elapsed = 0 ;
  hipEventElapsedTime(&elapsed, start, stop) ;
  fprintf(stderr, "It takes %lf millisecs\n", elapsed) ;
  hipMemcpy(a, a_gpu, SIZEX*SIZEY*sizeof(float), hipMemcpyDeviceToHost) ;
  hipMemcpy(b, b_gpu, SIZEX*SIZEY*sizeof(float), hipMemcpyDeviceToHost) ;
   FILE * f = fopen("result_cu", "w");
   dump_stencil(a, f) ;
   fclose(f) ;
  free(a);
  free(b);
  return 0;
}

